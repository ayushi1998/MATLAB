#include "hip/hip_runtime.h"
#include <stdio.h>
#include <complex>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
#include <mex.h>
#include "gpu/mxGPUArray.h"

#define M_PI 3.1415926535

using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}



__device__ double2 complexmul(const double2 a, const double2 b) {
	double2 res;
	res.x = a.x*b.x-a.y*b.y;
	res.y = a.x*b.y+a.y*b.x;
	return res;
}

__device__ double2 complexdiv(const double2 a, const double2 b) {
	double2 res;
	res.x = (a.x * b.x + a.y * b.y) / (b.x*b.x+b.y*b.y);
	res.y = (-a.x * b.y + a.y * b.x) / (b.x*b.x+b.y*b.y);
	return res;
}

__device__ double2 complexlog(const double2 a) {
	double2 res;
	res.x = 0.5 * log(a.x*a.x + a.y*a.y);
	res.y = atan2(a.y, a.x);
	if (res.y + M_PI < 1e-9)
		res.y = M_PI;
	return res;
}

__global__ void
calcCGCoords(const double2* curve, const double2* pts, double2* coords, int M, int N) {

	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int i = blockDim.x * blockIdx.x + threadIdx.x;	

    if (i >= M || j >= N)
        return;

    double2 Aj, Ajp1, Bj, Bjp1, Bjm1;

	Aj.x = curve[j].x - curve[(j + N - 1) % N].x;
	Aj.y = curve[j].y - curve[(j + N - 1) % N].y;

    Ajp1.x = curve[(j + 1) % N].x - curve[j].x;
    Ajp1.y = curve[(j + 1) % N].y - curve[j].y;

	Bj.x = curve[j].x - pts[i].x;
	Bj.y = curve[j].y - pts[i].y;

    Bjp1.x = curve[(j + 1) % N].x - pts[i].x;
	Bjp1.y = curve[(j + 1) % N].y - pts[i].y;

    Bjm1.x = curve[(j + N - 1) % N].x - pts[i].x;
	Bjm1.y = curve[(j + N - 1) % N].y - pts[i].y;

    
    double2 f1 = complexdiv(complexlog(complexdiv(Bj, Bjm1)), Aj);
    double2 f2 = complexdiv(complexlog(complexdiv(Bjp1, Bj)), Ajp1);
    
    coords[j * M + i].x = (f1.y-f2.y) / (2*M_PI);
    coords[j * M + i].y = -(f1.x-f2.x) / (2*M_PI);

}


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    
    mxInitGPU();
    
    mxGPUArray const * A = mxGPUCreateFromMxArray(prhs[0]);
    mwSize n = mxGPUGetNumberOfElements(A);
    double2 const * d_A = (double2 const *)(mxGPUGetDataReadOnly(A));
    
    mxGPUArray const * B = mxGPUCreateFromMxArray(prhs[1]);
    mwSize m = mxGPUGetNumberOfElements(B);
    double2 const * d_B = (double2 const *)(mxGPUGetDataReadOnly(B));
    
    mwSize dims[2] = {m, n};
    mxGPUArray* C = mxGPUCreateGPUArray(2, dims, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    double2* d_C = (double2*)(mxGPUGetData(C));
    
    dim3 block(32,32);
    dim3 grid((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);
    
    calcCGCoords<<<grid, block>>>(d_A, d_B, d_C, m, n);
    
    plhs[0] = mxGPUCreateMxArrayOnGPU(C);
    
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(C);
    
}